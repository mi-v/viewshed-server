#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>
#include "cuvshed.h"

#define sindf(a) sinpif((a) / 180)
#define cosdf(a) cospif((a) / 180)

#define cuErr(call) {if (hipSuccess != (call)) throw cuErrX{hipGetErrorString(hipGetLastError()), __LINE__};}

__constant__ float CUTOFF;
__constant__ float CUTON;
__constant__ int MAXZOOM;
__constant__ float DSTEP;
static Config config;

__device__ float interp(float a, float b, float f)
{
    return a + f * (b - a);
}

__device__ float seaDistR(LL p0, LL p1)
{
    LL d = p1 - p0;

    return 2 * asinf(sqrtf(
        sinf(d.lat/2) * sinf(d.lat/2) + cosf(p0.lat) * cosf(p1.lat) * sinf(d.lon/2) * sinf(d.lon/2)
    ));
}

__device__ float hgtQuery(const short** __restrict__ HgtMap, Recti rect, LL ll)
{
    //LLi lli = ll;
    //LLi lliof = lli - rect.ll;
    LLi lliof = LLi(ll) - rect.ll;
    const short* hgtCell = HgtMap[lliof.lat * rect.width + lliof.lon];
    if (!hgtCell) return 0;

    ll -= ll.floor();

    float Y = ll.lat * 1200;
    float X = ll.lon * 1200;

    int Xi = floorf(X);
    int Yi = floorf(Y);

    float Xf = X - Xi;
    float Yf = Y - Yi;

    int ofs = 1200 * 1280 - Yi * 1280 + Xi;
    float a = hgtCell[ofs];
    float b = hgtCell[ofs + 1];
    float c = hgtCell[ofs - 1280];
    float d = hgtCell[ofs - 1280 + 1];

    return (a * (1 - Xf) + b * Xf) * (1 - Yf) + (c * (1 - Xf) + d * Xf) * Yf;
}

__global__ void Query(const short** __restrict__ HgtMap, Recti rect, LL ll, float* result) {
    if (blockIdx.x || blockIdx.y || threadIdx.x || threadIdx.y) return;
    *result = hgtQuery(HgtMap, rect, ll);
}

__global__ void doScape(const short** __restrict__ HgtMap, Recti hgtRect, float* __restrict__ AzEleD, Vec3 myP, float myH, LL myL)
{
    int az = blockIdx.x * blockDim.x + threadIdx.x;
    int distN = blockIdx.y * blockDim.y + threadIdx.y;
    float dist = CUTON + DSTEP * distN * (distN + 1) / 2;
    float rDist = dist / ERAD;

    float azR = 2 * PI * az / ANGSTEPS;

    LL myR = myL.toRad();
    LL ptR = {asinf(sindf(myL.lat) * cosf(rDist) + cosdf(myL.lat) * sinf(rDist) * cosf(azR))}; // <- lat only! lon follows
    ptR.lon = myR.lon + atan2f(sinf(azR) * sinf(rDist) * cosdf(myL.lat), cosf(rDist) - sindf(myL.lat) * sinf(ptR.lat));

    LL ptL = ptR.fromRad();

    float hgt = hgtQuery(HgtMap, hgtRect, ptL);

    Vec3 ptP = (ERAD+hgt) * Vec3(ptR);

    Vec3 losP = myP - ptP;

    float trueDist = float(losP);

    float elev = 1 - myP * losP / (trueDist * (ERAD+myH));

    int ofs = distN * ANGSTEPS + az;
    AzEleD[ofs] = elev;
}

__global__ void elevProject(float* AzEleD)
{
    int az = blockIdx.x * blockDim.x + threadIdx.x;
    float elev = -1;
    for (int distN = 1; distN < DSTEPS; distN++) {
        int ofs = distN * ANGSTEPS + az;
        if (AzEleD[ofs] < elev) {
            AzEleD[ofs] = elev;
        } else {
            elev = AzEleD[ofs];
        }
    }
}

__global__ void doVisMap(
    const short** __restrict__ HgtMap,
    Recti hgtRect,
    const float* __restrict__ AzEleD,
    Vec3 myP,
    float myH,
    LL myL,
    Px2 pxBase,
    unsigned char* __restrict__ visMap
)
{
    Px2 imgPx = {
        int(blockIdx.x * blockDim.x + threadIdx.x),
        int(blockIdx.y * blockDim.y + threadIdx.y)
    };
    int visMapWidth = blockDim.x * gridDim.x;

    Px2 ptPx = pxBase + imgPx;

    LL ptR = ptPx.toLL(MAXZOOM);

    LL ptL = ptR.fromRad();

    float hgt = hgtQuery(HgtMap, hgtRect, ptL);

    Vec3 ptP = (ERAD+hgt) * Vec3(ptR);
    Vec3 losP = myP - ptP;

    float trueDist = float(losP);

    float elev = 1 - myP * losP / (trueDist * (ERAD+myH));

    LL myR = myL.toRad();

    float dist = ERAD * seaDistR(myR, ptR);
    int distN = floorf((sqrtf(1 + 8 * (dist - CUTON) / DSTEP) - 1) / 2);
    float distNdist = CUTON + DSTEP * distN * (distN + 1) / 2;

    float azR = atan2f(sinf(ptR.lon - myR.lon) * cosf(ptR.lat), cosf(myR.lat) * sinf(ptR.lat) - sinf(myR.lat) * cosf(ptR.lat) * cosf(ptR.lon - myR.lon));
    if (azR < 0) {
        azR += 2 * PI;
    }
    float azi;
    float azf = modff(ANGSTEPS * azR / (2 * PI), &azi);
    int az = azi;

    //visMap[visMapOffset] = 0;

    //if (distN >= DSTEPS) {
    /*if (__all_sync(~0, distN >= DSTEPS)) {
        //visMap[visMapOffset] = 1;
        return;
    }*/
    bool visible = false;

    if (distN < DSTEPS && elev + 0.0001 > interp(AzEleD[distN * ANGSTEPS + az], AzEleD[distN * ANGSTEPS + (az+1) % ANGSTEPS], azf)) {
        Px2 myPx = myR.toPx2(MAXZOOM);

        float pxDist = float(ptPx - myPx);

        LL llStep = (ptL - myL) / pxDist;

        float distStep = dist / pxDist;

        visible = true;
        int i = 10;
        while (dist > distNdist && i--) {
            dist -= distStep;
            ptL -= llStep;
            ptR = ptL.toRad();

            hgt = hgtQuery(HgtMap, hgtRect, ptL);

            Vec3 ptP = (ERAD+hgt) * Vec3(ptR);
            Vec3 losP = myP - ptP;

            trueDist = float(losP);

            float stepElev = 1 - myP * losP / (trueDist * (ERAD+myH));

            if (stepElev > elev) {
                visible = false;
                break;
            }
        }

        //visMap[visMapOffset] = 1;
    }

    int visMapOffset = visMapWidth * imgPx.y + imgPx.x;
    //visMap[visMapOffset] = visible ? 1 : 0;

    //unsigned char b = __brev(__ballot_sync(~0, visible)) >> (threadIdx.x % 32 - 24);
    unsigned bb = __brev(__ballot_sync(~0, visible));
    if (threadIdx.x % 8 == 0) {
        unsigned char b = bb >> (24 - threadIdx.x % 32);
        visMap[visMapOffset / 8] = b;
    }
}

extern "C" {
    float Query(const short** HgtMap, Recti rect, LL ll) {
        float* d_result;
        float result;
        hipMalloc((void**)&d_result, sizeof(float));
        Query<<<1, 1>>>(HgtMap, rect, ll, d_result);
        int r = hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_result);
        return result;
    }

    Image makeImage(LL myL, int myH, const uint64_t* HgtMapIn, Recti hgtRect) {
        const short** HgtMap = NULL;
        float* AzEleD = NULL;
        unsigned char* Img_d = NULL;
        Image Img = {};

        try {
            cuErr(hipMalloc(&HgtMap, hgtRect.width * hgtRect.height * sizeof(uint64_t)));
            cuErr(hipMemcpy(HgtMap, HgtMapIn, hgtRect.width * hgtRect.height * sizeof(uint64_t), hipMemcpyHostToDevice));

            LL myR = myL.toRad();
            Vec3 myP = (ERAD + Query(HgtMap, hgtRect, myL) + myH) * Vec3(myR);

            cuErr(hipMalloc(&AzEleD, ANGSTEPS * DSTEPS * sizeof(float)));

            doScape<<<dim3(ANGSTEPS/256, DSTEPS), dim3(256, 1)>>>(
                HgtMap,
                hgtRect,
                AzEleD,
                myP,
                myH,
                myL
            );
            cuErr(hipGetLastError());

            elevProject<<<dim3(ANGSTEPS/32), dim3(32)>>>(AzEleD);
            cuErr(hipGetLastError());

            LL rngR = {config.CUTOFF / ERAD};
            rngR.lon = -rngR.lat / cosf(myR.lat);

            Img.rect.P = (myR + rngR).toPx2(config.MAXZOOM);
            Img.rect.P.x &= ~255;
            Img.rect.P.y &= ~255;
            Img.rect.Q = (myR - rngR).toPx2(config.MAXZOOM);
            Img.rect.Q.x |= 255;
            Img.rect.Q.y |= 255;
            Img.rect.Q.x += 1;
            Img.rect.Q.y += 1;

            cuErr(hipMalloc(&Img_d, Img.wh() / 8));

            doVisMap<<<dim3(Img.w()/256, Img.h()), dim3(256, 1)>>>(
                HgtMap,
                hgtRect,
                AzEleD,
                myP,
                myH,
                myL,
                Img.rect.P,
                Img_d
            );
            cuErr(hipGetLastError());

            Img.buf = malloc(Img.wh());
            cuErr(hipMemcpy(Img.buf, Img_d, Img.wh() / 8, hipMemcpyDeviceToHost));
        } catch (cuErrX error) {
            Img.error = error;
        }

        hipFree(HgtMap);
        hipFree(AzEleD);
        hipFree(Img_d);
        return Img;
    }

    void Init(Config c) {
        hipMemcpyToSymbol(HIP_SYMBOL(CUTOFF), &c.CUTOFF, sizeof(CUTOFF));
        hipMemcpyToSymbol(HIP_SYMBOL(CUTON), &c.CUTON, sizeof(CUTON));
        hipMemcpyToSymbol(HIP_SYMBOL(MAXZOOM), &c.MAXZOOM, sizeof(MAXZOOM));
        float dstep = 2 * (c.CUTOFF - c.CUTON) / (DSTEPS * (DSTEPS - 1));
        hipMemcpyToSymbol(HIP_SYMBOL(DSTEP), &dstep, sizeof(DSTEP));
        config = c;
    }

    void stopprof() {
        hipProfilerStop();
    }
}
